#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <windows.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include "gl\glew.h"
#include <gl\GL.h>
#include <gl\glu.h>
#include "gl\glut.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <cuda_gl_interop.h>

#include "opencv2\highgui\highgui.hpp"
#include "opencv2\imgproc\imgproc.hpp"

#define GET_PROC_ADDRESS(str) wglGetProcAddress(str)
#define DIM 128

PFNGLBINDBUFFERARBPROC    glBindBuffer = NULL;
PFNGLDELETEBUFFERSARBPROC glDeleteBuffers = NULL;
PFNGLGENBUFFERSARBPROC    glGenBuffers = NULL;
PFNGLBUFFERDATAARBPROC    glBufferData = NULL;

GLuint bufferObj;
hipGraphicsResource *resource;
GLuint bufferObj1;
hipGraphicsResource *resource1;

const GLfloat lightPosition[] = {10.0,10.0,10.0,0.0};
const GLfloat whiteLight[] = {0.8,0.8,0.8,1.0};
GLfloat matSpecular [] = {0.3,0.3,0.3,1.0};
GLfloat matShininess [] = {20.0};
//GLfloat matEmission [] = {0.3,0.3,0.3,1.0};
GLfloat matEmission [] = {1,1,1,1.0};
GLfloat matEmission2 [] = {0,0,0,0.0};
GLfloat spin = 180;
GLfloat projMatrix[16] = {  1.1160 ,   1.9906   , 0.0373 ,   0.1244,
	1.3925  , -1.4027  ,  0.1231  ,  0.4105,
	-0.8277 ,  0.3304 ,   0.2710  ,  0.9033,
	0.3545 ,  -0.2735 ,   1.3939 ,   4.6464  };
GLfloat projMatrix1[16] = {  1.11599 ,   1.99059   , 0.0373221 ,   0.124407,
	1.39251  , -1.40265  ,  0.123144  ,  0.41048,
	-0.827706 ,  0.330419 ,   0.271003  ,  0.903343,
	0.354539 ,  -0.273485 ,   1.39393 ,   4.64644  };
GLfloat headTran[16] = { 2.04317, 2.88461 ,-0.0186038 ,0, 
	-3.00478, 2.0353 ,-0.293618,0 ,
	0.232629 ,-0.180116, -3.62916 ,0,
	1.79757 ,-1.13934 ,-0.917011, 1 };
GLfloat mProj[16];

bool flag = false;
void grab()   
{  
	int width = 320;
	int heith = 256;
	char *mpixels = new  char[width * heith * 3];
	glReadBuffer(GL_FRONT);
	glReadPixels(0, 0, width, heith, GL_RGB, GL_BYTE, mpixels);
	//glReadBuffer(GL_BACK);
	for(int i = 0;i < (int)width*heith*3;i += 3)  
	{     
		mpixels[i] ^= mpixels[i+2] ^= mpixels[i] ^= mpixels[i+2];
	}  
	cv::Mat dispImg;
	dispImg.create(cv::Size(width,heith), CV_8UC3 );
	for(int y = 0 ;y < heith;y++)  
	{  
		for(int x = 0 ;x < width;x++)  
		{  
			dispImg.at<cv::Vec3b>(y,x)[0] =mpixels[(y*width+x) * 3 + 0];
			dispImg.at<cv::Vec3b>(y,x)[1] =mpixels[(y*width+x) * 3 + 1];
			dispImg.at<cv::Vec3b>(y,x)[2] =mpixels[(y*width+x) * 3 + 2];
		}  

	}  
	cv::Mat dst;
	//32 BIT - 32 BIT
	cvtColor(dispImg, dst, CV_RGB2GRAY);
	width = dst.cols;
	int heigh = dst.rows;
	for ( int x=0;x<width/2 ;x++)
	{
		for ( int y=0;y<heigh/2;y++)
		{
			dst.at<uchar>(y,x) = 0;
			continue;
		}
	}
	cv::imwrite( "test.jpg", dst );
}
void init()
{
	// Add lighting here
	// Add lighting here
	GLfloat ambient[]  = {(float)0.4, (float)0.4, (float)0.4, (float)1.0};
	GLfloat diffuse[]  = {(float)1.0, (float)1.0, (float)1.0, (float)1.0};
	GLfloat specular[] = {(float)1.0, (float)1.0, (float)1.0, (float)1.0};
	GLfloat lightpos[] = {0.0f,0.0f,1.0f,0.0f};
	// Init all the lighting
	glLightfv(GL_LIGHT0, GL_AMBIENT, ambient);
	glLightfv(GL_LIGHT0, GL_SPECULAR, specular);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, diffuse);
	glLightfv(GL_LIGHTING,GL_POSITION,lightpos);

	//glEnable(GL_LIGHT0);
	glEnable(GL_LIGHTING);
	// Add basic camera here

	glClearColor(0.0f,0.0f,0.0f,1.0f);
	glClearDepth(1.0);
	//glClearColor(1.0f,1.0f,1.0f,1.0f);
	glCullFace(GL_BACK);
	glFrontFace(GL_CCW);

	//glClearColor(1.0f,1.0f,1.0f,1.0f);
	glClearStencil(0);
	glEnable(GL_DEPTH_TEST);
	glDepthFunc(GL_LEQUAL);	
	glEnable(GL_NORMALIZE);
	glEnable(GL_AUTO_NORMAL);

	glEnable(GL_LINE_SMOOTH);
	glEnable(GL_POINT_SMOOTH);
	glEnable(GL_POLYGON_SMOOTH);
	glEnable(GL_COLOR_MATERIAL);
	glHint(GL_POINT_SMOOTH_HINT, GL_NICEST);
	glHint(GL_LINE_SMOOTH_HINT, GL_NICEST);
	glHint(GL_POLYGON_SMOOTH_HINT, GL_NICEST);
	glHint(GL_PERSPECTIVE_CORRECTION_HINT, GL_NICEST);

	glShadeModel(GL_SMOOTH);
}
void display()  
{
	if(flag)
	{
		glDrawPixels(DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0);
		glutSwapBuffers();
	}
	else
	{
		GLint    viewport[4];
		GLdouble modelview[16];
		GLdouble projection[16];
		//GLdouble  winX, winY, winZ;
		GLdouble posX, posY, posZ;
		GLdouble posX1, posY1, posZ1;

		posX = -0.4054;	posY = 0.2393;	posZ = -2.5986;
		posX = -0.0;	posY = -0;		posZ =  -20.0;
		posX = -0.2421;	posY = 0.4427;	posZ = 0.2279;
		posX1 = 0.3052;	posY1 = 0.4601;	posZ1 = 0.2360;
	
		glClear(GL_COLOR_BUFFER_BIT|GL_DEPTH_BUFFER_BIT);
		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		glMultMatrixf(projMatrix);
		glMatrixMode(GL_MODELVIEW);
		glLoadIdentity();
		glMultMatrixf(headTran);
		glPushMatrix();
			glTranslatef(posX, posY, posZ);
			//std::cout<<spin<<std::endl;
				glRotatef(spin,0.0,1.0,0.0);
				glMaterialfv(GL_FRONT,GL_SPECULAR, matSpecular);
				glMaterialfv(GL_FRONT,GL_SHININESS, matShininess);
				glMaterialfv(GL_FRONT,GL_EMISSION, matEmission);
				glutSolidSphere(0.1507,50,50);
				glPushMatrix();
				glTranslatef(0.1507*0.8, 0, 0);
				glMaterialfv(GL_FRONT,GL_EMISSION, matEmission2);
				//glColor3f(0.0f, 0.0f, 0.0f);
				glutSolidSphere(0.4*0.1507,50,50);
				glPopMatrix();
			glPopMatrix();
		glPushMatrix();
		glutSwapBuffers();
		glFlush();
		//flag = true;
	}
}
__global__ void cudaGLKernel(uchar4 *ptr, uchar4 *ptr1)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
      
    /*float fx = x / (float)DIM - 0.5f;
    float fy = y / (float)DIM - 0.5f;*/  
      
    ptr[offset].x = ptr1[offset].x;
    ptr[offset].y = ptr1[offset].y;
    ptr[offset].z = ptr1[offset].z;
    ptr[offset].w = ptr1[offset].w;
}
__global__ void cudaGray(uchar4 *ptr, uchar4 *ptr1, float *result, float *result1)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    /*float fx = x / (float)DIM - 0.5f;
    float fy = y / (float)DIM - 0.5f;*/  

	result[offset] = ptr[offset].x * 0.3 +ptr[offset].y * 0.3 + ptr[offset].z * 0.4;
	result1[offset] = ptr1[offset].x * 0.3 +ptr1[offset].y * 0.3 + ptr1[offset].z * 0.4;
}
__global__ void cudaCaculate(float *result, float *result1, double *answer)
{
    /*float fx = x / (float)DIM - 0.5f;
    float fy = y / (float)DIM - 0.5f;*/
	int length = DIM * DIM;
	int z = 0;
	answer[0]=0;
	answer[1]=0;
	answer[2]=0;
	answer[3]=0;
	answer[4]=0;
	int offset = 0;
	for (int x = 0; x < DIM; x++)
	{
		for (int y = 0; y < DIM; y++)
		{
			answer[0] += result[offset];//SumX
			answer[1] += result1[offset];//SumY
			answer[2] += (result[offset]*result[offset]);//SumXX
			answer[3] += (result[offset]*result1[offset]);//SumXY
			answer[4] += (result1[offset]*result1[offset]);//SumYY
			offset++;
		}
	}
	answer[5] = (DIM*DIM*answer[3] - answer[0]*answer[1]) / (sqrt(DIM*DIM*answer[2]-answer[0]*answer[0])*sqrt(DIM*DIM*answer[4]-answer[1]*answer[1]));
}
void drawFunc(void)
{
	glReadBuffer(GL_FRONT);
	glReadPixels(0, 0, DIM, DIM, GL_BGRA, GL_UNSIGNED_BYTE, 0);
	//CUDAӳ�乲����Դ������ȡ������Կ����Ե��豸ָ��
    uchar4* devPtr;
    size_t size;
    hipGraphicsMapResources(1, &resource, NULL);
    hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, resource);
	
	uchar4* devPtr1;
    size_t size1;
    hipGraphicsMapResources(1, &resource1, NULL);
    hipGraphicsResourceGetMappedPointer((void**)&devPtr1, &size1, resource1);

	//ִ��CUDA�˺���
    
	if(flag == false)
	{
		dim3 grids(DIM / 16, DIM / 16);
		dim3 threads(16, 16);
		cudaGLKernel<<<grids, threads>>>(devPtr, devPtr1);
		printf("\nsave picture to devPtr");
	}
	else
	{
		hipError_t cudaStatus;
		printf("\ncalculating");
		float *result, *result1;
		double *answer, *cpuresult;
		dim3 grids(DIM / 16, DIM / 16);
		dim3 threads(16, 16);
		size_t nBytes = DIM * DIM * sizeof(float);
		size_t nBytes1 = 6 * sizeof(double);
		cudaStatus = hipMalloc((float **)&result, nBytes);
		if (cudaStatus != hipSuccess) {  
			fprintf(stderr, "hipMalloc failed!");  
			return;  
		}
		cudaStatus = hipMalloc((float **)&result1, nBytes);
		if (cudaStatus != hipSuccess) {  
			fprintf(stderr, "hipMalloc failed!");
			hipFree(result);
			return;  
		}
		cudaStatus = hipMalloc((double **)&answer, nBytes1);
		if (cudaStatus != hipSuccess) {  
			fprintf(stderr, "hipMalloc failed!");
			hipFree(result);
			hipFree(result1);
			return;  
		}
		cpuresult = (double *)malloc(nBytes1);
		//�������һ�»Ҷȣ����ƶ�����
		cudaGray<<<grids, threads>>>(devPtr, devPtr1, result, result1);
		//�ȴ����
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {  
			printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n");  
			hipFree(result);
			hipFree(result1);
			hipFree(answer);
			free(cpuresult);
			return;
		}
		//���̼߳������ϵ��
		cudaCaculate<<<1, 1>>>(result, result1, answer);
		//�ȴ����
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {  
			printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n");  
			hipFree(result);
			hipFree(result1);
			hipFree(answer);
			free(cpuresult);
			return;
		}
		//�ѽ�����Դ渴�Ƶ��ڴ�
		cudaStatus = hipMemcpy(cpuresult, answer, nBytes1, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {  
			printf("hipMemcpy failed!");
			hipFree(result);
			hipFree(result1);
			hipFree(answer);
			free(cpuresult);
			return;
		}
		printf("\nSumX:%lg \nSumY:%lg \nSumXX:%lg \n%SumXY:lg \n%SumYY:lg \nResult:%lg", cpuresult[0], cpuresult[1], cpuresult[2], cpuresult[3], cpuresult[4], cpuresult[5]);
		hipFree(result);
		hipFree(result1);
		hipFree(answer);
		free(cpuresult);
		flag = false;
	}
	//���CUDA�Թ���������ӳ��
	//��������ӳ�䣬OpenGL��û��Ȩ�޷��ʹ���������
    hipGraphicsUnmapResources(1, &resource, NULL);
	hipGraphicsUnmapResources(1, &resource1, NULL);
}
void reshape(int w,int h)  
{
	glViewport(0.0,0.0,(GLsizei) w,(GLsizei) h);
}
void keyboardFunc(unsigned char key,int x,int y)
{
	switch(key)  
	{
	case 'a':  
		spin +=10;
		flag = false;
		break;
	case 'd':  
		spin -=10;
		flag = false;
		break;
	case 's':
		drawFunc();
		break;
	case 'f':
		clock_t start, finish, end;
		start = clock();
		for(int i = 0; i < 900; i++)
		{
			drawFunc();
		}
		finish = clock();
		for(int i = 0; i < 900; i++)
		{
			grab();
		}
		end = clock();
		printf("\n%d\n%d", finish-start, end-finish);
		break;
	case 'q':
		flag = true;
		drawFunc();
		break;
	case 'g':
		flag = true;
		break;
	}
	//if(spin<360)  
	//	spin +=360;
	//else if(spin>=360)  
	//	spin -=360;
	glutPostRedisplay();
}
void mouseFunc(int button,int state,int x,int y)
{  
	if(state == GLUT_DOWN){  
		/*switch (button)  
		{  
		case GLUT_LEFT_BUTTON:  
			matEmission[0]+=1.0;
			if(matEmission[0]>1.0)  
				matEmission[0]-=1.1;
			break;
		case GLUT_MIDDLE_BUTTON:  
			matEmission[1]+=1.0;
			if(matEmission[1]>1.0)  
				matEmission[1]-=1.1;
			break;
		case GLUT_RIGHT_BUTTON:  
			matEmission[2]+=1.0;
			if(matEmission[2]>1.0)  
				matEmission[2]-=1.1;
			break;
		}  */
		glutPostRedisplay();
	}
}
int main(int argc, char* argv[])
{
    //ѡ���ڼ��㹦�ܼ��İ汾��Ϊ1.0��GPU�豸������
	//CUDA5.5���Ժ�İ汾��������
    //hipDeviceProp_t prop;
    //int dev;
    //memset(&prop, 0, sizeof(hipDeviceProp_t));
    //prop.major = 1;
    //prop.minor = 0;
    //hipChooseDevice(&dev, &prop);
	//hipError_t cudaStatus;
    //cudaStatus = cudaGLSetGLDevice(dev);
    //if (cudaStatus != hipSuccess) {  
    //    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
    //    return 0;
    //}

    //OpenGL������ʼ��    
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(DIM, DIM);
    glutCreateWindow("CUDA + OpenGL");
      
    glBindBuffer = (PFNGLBINDBUFFERARBPROC)GET_PROC_ADDRESS("glBindBuffer");
    glDeleteBuffers = (PFNGLDELETEBUFFERSARBPROC)GET_PROC_ADDRESS("glDeleteBuffers");
    glGenBuffers = (PFNGLGENBUFFERSARBPROC)GET_PROC_ADDRESS("glGenBuffers");
    glBufferData = (PFNGLBUFFERDATAARBPROC)GET_PROC_ADDRESS("glBufferData");

	//����һ�����������, ������󶨵����ػ�����(����������ŵ���������Ϊ��PBO)
	//�����Կ��ڴ�ռ䲢������������Լ���ʼֵ
	//GL_ARRAY_BUFFER 3D��ͼ
	//GL_PIXEL_UNPACK_BUFFER_ARB ƽ���ͼ
	//���������ͻ��кܶࡣ����
    glGenBuffers(1, &bufferObj);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, DIM*DIM * 16, NULL, GL_DYNAMIC_DRAW_ARB);

	//GL_PIXEL_UNPACK_BUFFER_ARB GL_PIXEL_PACK_BUFFER_ARB����
	//unpack���ӻ�������opengl�����ú�glDrawPixel��glTexImage2D��glTexSubImage2D��Ĭ����Դ
	//pack����opengl�������������ú�glReadPixel��glGetTexImage��Ĭ��Ŀ��
    glGenBuffers(1, &bufferObj1);
    glBindBuffer(GL_PIXEL_PACK_BUFFER_ARB, bufferObj1);
    glBufferData(GL_PIXEL_PACK_BUFFER_ARB, DIM*DIM * 16, NULL, GL_DYNAMIC_DRAW_ARB);

	//�ѻ�������CUDA
	//cudaGraphicsMapFlagsNone
	//hipGraphicsRegisterFlagsReadOnly
	//hipGraphicsRegisterFlagsWriteDiscard
	hipGraphicsGLRegisterBuffer(&resource, bufferObj, cudaGraphicsMapFlagsNone);
	hipGraphicsGLRegisterBuffer(&resource1, bufferObj1, cudaGraphicsMapFlagsNone);

	//Opengl��ʾͼƬ
	glutDisplayFunc(display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(keyboardFunc);
	glutMouseFunc(mouseFunc);
	init();
    glutMainLoop();
    return 0;
}